
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>


long long getFileSize(char *name){

    FILE *file;
    long long sz = 0;
    file = fopen(name, "r");
    if (file == NULL)
    {
        printf("File is not available \n");
        return 0;
    }
    else{
	long long prev=ftell(file);
        fseek(file, 0L, SEEK_END);
        sz=ftell(file);
	fseek(file,prev,SEEK_SET);
    }
    fclose(file);
    return sz;
}

__global__ void search(char* data, int data_len, char* pat, int pat_len, int offset, int* c)
{
   int tid = blockIdx.x * blockDim.x + threadIdx.x;
   //printf("ThreadId: %d\n", tid);
   int M = pat_len;
   int N = (tid + 1) * offset;
   //printf("\ntM: %d\t\tN: %d\t\tstart:%d\t\tM+N: %d", M, N, tid*offset, M+N);
   //printf("\nstart index: %d\t\tend index: %d", tid*offset, N + M);
   if(tid*offset <= data_len){
           //printf("\ndatalen: %d", data_len);
           //printf("\nM: %d\t\tN: %d\t\tstart:%d\t\tM+N: %d", M, N, tid*offset, M+N);
	   //printf("\nstart index: %d\t\tend index: %d", tid*offset, N + M);
	   int i, j;
	   int count = 0;
	   int found = 0;
	   for(i = tid*offset; i < N + 1; i++) {
	     found = 1;
	     for(j = 0; j < M; j++) {
               //printf("%d\t", i+j);
	       if(data[i+j] != pat[j]) {
		 found = 0;
		 break;
	       }
	   }
	   //printf("\n%daaaa", N*(M-1));
	     if(found) {
	       //atomicAdd(&count, 1);
	       count++;
	       i = i + M -1;
	       //printf("at index: %d\n",i);
	       //atomicAdd(&count, 1);
	     }
	   }
	   
	   //printf("\n%d  ",count);
	   //cuPrintf()
	   //*c += count;
	   atomicAdd(c, int(count));
           //printf("%d", c);
    }
}


int main(){

    	clock_t t;
    	t = clock();

	long long size = getFileSize("largeData.txt");
	printf("%lld", size);
	long long i = 0;
	char *data = (char *)malloc(size*sizeof(char));
	char ch;


	FILE *file;
	file = fopen("largeData.txt", "r");
	if (file == NULL){
		printf("File is not available \n");
	}
	else{
		while ((ch = fgetc(file)) != EOF){
			data[i] = ch;
			i++;
		}
	}


	int count;
	char pat[100] = "Angina  n. (in full angina pectoris) chest pain brought on by exertion, caused by an inadequate bl";
	int pat_s = strlen(pat);
	
	char *dev_data;
	char *dev_pat;
	int *dev_count;

	hipMalloc((void**)&dev_data, size*sizeof(char));
	hipMalloc((void**)&dev_pat, pat_s*sizeof(char));
	hipMalloc((void **)&dev_count, sizeof(int));

	hipMemcpy(dev_data, data, size*sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(dev_pat, pat, pat_s*sizeof(char), hipMemcpyHostToDevice);

        int threads_per_block = 150;
        int block_size = 100;
        int offset = (size/(block_size * threads_per_block)) + 1;


	search<<<block_size,threads_per_block>>>(dev_data, size, dev_pat, pat_s, offset, dev_count);


	hipMemcpy(&count, dev_count, sizeof(int), hipMemcpyDeviceToHost);
	printf("\nfrequency of \"%s\" in data: %d  ",pat, count);
	fclose(file);
	
	hipFree(dev_data);
	hipFree(dev_pat);
	hipFree(dev_count);

	t = clock() - t;
	double time_taken = ((double)t)/CLOCKS_PER_SEC;
	printf("\nIt took %f seconds for parallel code to execute \n", time_taken);

	return 0;
}

