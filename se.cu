
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>

long long getFileSize(char *name){

	FILE *file;
	file = fopen(name, "r");
	//printf("2\n");
	if (file == NULL)
    {
        printf("File is not available \n");
        return 0;
    }
    long long prev=ftell(file);
    //printf("3\n");
    fseek(file, 0L, SEEK_END);
    //printf("4\n");
    long long sz=ftell(file);
    //printf("5\n");
    fseek(file,prev,SEEK_SET);
    //printf("6\n"); //go back to where we were
    fclose(file);
    //printf("7\n");
    return sz;
}

void search(char* data, int data_len, char* pat, int pat_len, int offset)
{
   int M = pat_len;
   int N = offset;
   //printf("%s", data);
   int i, j;
   int count = 0;
   int found = 0;
   for(i = 0; i < N - M + 1 ; i++) {
     found = 1;
     for(j = 0; j < M; j++) {
       if(data[i+j] != pat[j]) {
         found = 0;
         break;
       }
     }
     if(found) {
       count++;
       i = i + M -1;
       //printf("at index: %d\n",i);
     }
   }
   printf("frequency of \"%s\" in data: %d  ",pat, count);
}

int main(){

    clock_t t;
    t = clock();

	FILE *file;
	file = fopen("largeData.txt", "r");
	//printf("1\n");
	long long size = getFileSize("largeData.txt");
	//printf("8\n");

	//printf("%d\n", 1000000);
	long long i = 0;
	char *data = (char*)malloc(size*sizeof(char));
	//printf("9\n");
    char ch;
    if (file == NULL)
    {
        printf("File is not available \n");
    }
    else
    {
        //printf("10\n");
        while ((ch = fgetc(file)) != EOF)
        {
            data[i] = ch;
            i++;
        }
        //printf("11\n");
    }

    //for(long long j=0;j<size;j++)
    	//printf("%c", data[j]);
    //printf("12\n");
    char pat[100] = "Angina  n. (in full angina pectoris) chest pain brought on by exertion, caused by an inadequate bl";
    int pat_s = strlen(pat);
    search(data, size, pat, pat_s, size);
    fclose(file);
    //printf("13\n");

    t = clock() - t;
    double time_taken = ((double)t)/CLOCKS_PER_SEC; // in seconds

    printf("\nIt took %f seconds for serial code to execute \n", time_taken);

    return 0;
}

